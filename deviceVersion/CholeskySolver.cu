#include "header.cuh"


/* Solve linear equation AX = B with Cholesky factorization */
void choleskySolver(hipComplex* d_X, hipComplex* d_A, hipComplex* d_B,
	int lda, int ldb, int m) {
	hipsolverHandle_t handle = NULL;
	hipStream_t stream = NULL;
	

	hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
	hipError_t cudaStat1 = hipSuccess;
	hipError_t cudaStat2 = hipSuccess;
	hipError_t cudaStat3 = hipSuccess;
	hipError_t cudaStat4 = hipSuccess;

	const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
	/*const int lda = 3;
	const int ldb = 3;
	const int m = 3;*/
	/* Create handle*/
	status = hipsolverDnCreate(&handle);
	assert(HIPSOLVER_STATUS_SUCCESS == status);

	cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
	assert(hipSuccess == cudaStat1);

	status = hipsolverSetStream(handle, stream);
	assert(status == HIPSOLVER_STATUS_SUCCESS);
	/*
	Initialize test matrices
	hipComplex A0[lda*m]; // = { 1.0, 2.0, 3.0, 2.0, 5.0, 5.0, 3.0, 5.0, 12.0 };
	A0[0] = make_hipComplex(1, 0);
	A0[1] = make_hipComplex(2, 0);
	A0[2] = make_hipComplex(3, 0);
	A0[3] = make_hipComplex(2, 0);
	A0[4] = make_hipComplex(5, 0);
	A0[5] = make_hipComplex(5, 0);
	A0[6] = make_hipComplex(3, 0);
	A0[7] = make_hipComplex(5, 0);
	A0[8] = make_hipComplex(12, 0);

	hipComplex B0[m] = { 1.0, 1.0, 1.0 };
	hipComplex X0[m]; // X0 = A0\B0 

	hipComplex L0[lda*m]; // cholesky factor of A0 

	printf("A0 = (matlab base-1)\n");
	printMatrix(m, m, A0, lda, "A0");
	printf("=====\n");

	printf("B0 = (matlab base-1)\n");
	printMatrix(m, 1, B0, ldb, "B0");
	printf("=====\n");
	*/

	//hipComplex* d_A = NULL;
	//hipComplex* d_B = NULL;
	int* d_info = NULL;

	/* Copy data to device*/
	/*
	cudaStat1 = hipMalloc((void**)&d_A, sizeof(hipComplex)*lda*m);
	cudaStat2 = hipMalloc((void**)&d_B, sizeof(hipComplex)*m);
	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);

	cudaStat1 = hipMemcpy(d_A, A0, sizeof(hipComplex)*lda*m, hipMemcpyHostToDevice);
	cudaStat2 = hipMemcpy(d_B, B0, sizeof(hipComplex)*m, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);
	*/

	cudaStat1 = hipMalloc((void**)&d_info, sizeof(int));
	assert(hipSuccess == cudaStat1);

	int Lwork = 0;
	/* Calculate the size of workspace*/
	status = hipsolverDnCpotrf_bufferSize(handle,
		uplo,
		m,
		d_A,
		lda,
		&Lwork);
	cudaStat1 = hipDeviceSynchronize();
	assert(HIPSOLVER_STATUS_SUCCESS == status);
	assert(hipSuccess == cudaStat1);

	hipComplex* workSpace;
	cudaStat1 = hipMalloc(&workSpace, sizeof(hipComplex)*Lwork);
	assert(hipSuccess == cudaStat1);

	/* Cholesky factorization*/
	status = hipsolverDnCpotrf(handle,
		uplo,
		m,
		d_A,
		lda,
		workSpace,
		Lwork,
		d_info);
	cudaStat1 = hipDeviceSynchronize();
	assert(HIPSOLVER_STATUS_SUCCESS == status);
	assert(hipSuccess == cudaStat1);

	
	/*cudaStat2 = hipMemcpy(L0, d_A, sizeof(hipComplex) * lda * m, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);

	printf("L = (matlab base-1), upper triangle is don't care \n");
	printMatrix(m, m, L0, lda, "L0");
	printf("=====\n"); */

	/* Solve equation*/
	status = hipsolverDnCpotrs(handle,
		uplo,
		m,
		1,
		d_A,
		lda,
		d_B,
		ldb,
		d_info);
	cudaStat1 = hipDeviceSynchronize();
	assert(HIPSOLVER_STATUS_SUCCESS == status);
	assert(hipSuccess == cudaStat1);


	cudaStat2 = hipMemcpy(d_X, d_B, sizeof(hipComplex)*m, hipMemcpyDeviceToDevice);
	
	assert(hipSuccess == cudaStat2);
	hipDeviceSynchronize();
	/*
	hipComplex h_X[3];
	cudaStat1 = hipMemcpy(h_X, d_X, sizeof(hipComplex)*m, hipMemcpyDeviceToHost);

	printMatrix(m, 1, h_X, ldb, "X");*/


	/*
	printf("X0 = (matlab base-1)\n");
	printMatrix(m, 1, d_X, ldb, "X0");
	printf("=====\n");
	*/
	/* free resources */
	//if (d_Aarray) hipFree(d_Aarray);
	//if (d_Barray) hipFree(d_Barray);
	//if (d_infoArray) hipFree(d_infoArray);

	if (handle) hipsolverDnDestroy(handle);
	if (stream) hipStreamDestroy(stream);

	return;
}