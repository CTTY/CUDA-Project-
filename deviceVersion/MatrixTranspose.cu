#include "header.cuh"

/* C = A^T*/
void transpose(hipComplex* C, hipComplex *A, int m, int n, int lda, int ldc)
{
	hipblasStatus_t stat; // CUBLAS functions status 
	hipblasHandle_t handle; // CUBLAS context 

	stat = hipblasCreate(&handle); // initialize CUBLAS context

	hipComplex alpha = make_hipComplex(1, 0);
	hipComplex beta = make_hipComplex(0, 0);

	hipComplex* temp;
	hipMalloc((void**)&temp, sizeof(hipComplex)*SIZE);
	int ldb = M*L;
	stat = hipblasCgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, &alpha, A, lda, &beta, temp, ldb, C, ldc);
	assert(stat == HIPBLAS_STATUS_SUCCESS);

	hipblasDestroy(handle); // destroy CUBLAS context
	if (temp) hipFree(temp);
}

/*C = A - B*/
void subtraction(hipComplex* C, hipComplex *A, hipComplex* B, int m, int n, int lda, int ldc) { 
	hipblasStatus_t stat; // CUBLAS functions status 
	hipblasHandle_t handle; // CUBLAS context 

	stat = hipblasCreate(&handle); // initialize CUBLAS context

	hipComplex alpha = make_hipComplex(1, 0);
	hipComplex beta = make_hipComplex(-1, 0);
	stat = hipblasCgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &alpha, A, lda, &beta, B, lda, C, ldc);
	assert(stat == HIPBLAS_STATUS_SUCCESS);

	hipblasDestroy(handle); // destroy CUBLAS context
}
