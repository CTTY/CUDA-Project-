#include "header.cuh"

class TestCreateCubeOnHost_1 {
public: 
	TestCreateCubeOnHost_1(hipComplex* h_cube) {
		createCubeOnHost(h_cube);
	}
};

class TestFFTCube_2 {
public:
	TestFFTCube_2(hipfftComplex* inputData, hipfftComplex* outputData, int BATCH, int DATASIZE) {
		batchFFT(inputData, outputData, BATCH, DATASIZE);
	}
};

class TestSteeringVector_3 {
public:
	TestSteeringVector_3(hipComplex* d_t) {
		steeringVector << <N, M >> > (d_t);
	}
};

class TestVectorizeSlices_4 {
public:
	TestVectorizeSlices_4(hipComplex* d_cube, hipComplex* d_y) {
		transpose(d_y, d_cube, M*L, N, N, M*L);
		//dim3 threads(M, N);
		//vectorizeSlices << <L, threads >> > (d_cube, d_y);
	}
};

class TestComputeS_5 {
public:
	TestComputeS_5(hipComplex* d_s, hipComplex* d_y) {
		/* Y * Y^H */
		matrixMulC(d_s, d_y, d_y, M*N, M*N, L, M*N, M*N, M*N, HIPBLAS_OP_N, HIPBLAS_OP_C);
	}
};

class TestConj_6 {
public:
	TestConj_6(hipComplex* d_t) {
		getConj << <1, M*N >> > (d_t);
	}
};

class TestQR_7 {
public:
	TestQR_7(hipComplex* d_s, int MN,
		hipComplex* t_conj,
		hipComplex* d_u) {
		choleskySolver(d_u, d_s, t_conj, M*N, M*N, M*N);
	}
};

class TestComputeTU_8 {
public:
	TestComputeTU_8(hipComplex* d_t, hipComplex* d_u, hipComplex* d_tempRes) {
		matrixMulC(d_tempRes, d_t, d_u, 1, L, M*N, M*N, M*N, 1, HIPBLAS_OP_C, HIPBLAS_OP_N);

	}
};

int main() {	

	std::ofstream outFile;
	std::string fileName = std::to_string(M) + "x" + std::to_string(N) + "x" + std::to_string(L) + ".txt";
	outFile.open(fileName);
	outFile << "Test begin!\n";
	outFile << "Test with cube size: M: "<< M <<", N: " << N <<", L: "<< L <<std::endl;
	hipEvent_t start0, stop0;
	float elapsedTime0;
	hipEventCreate(&start0); //Timing begins
	hipEventCreate(&stop0);
	hipEventRecord(start0, 0);

	/* Test 1: Create the cube*/
	hipEvent_t start, stop;
	float elapsedTime1;
	hipEventCreate(&start); //Timing begins
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipComplex *h_cube = new hipComplex[SIZE];

	TestCreateCubeOnHost_1* test1 = new TestCreateCubeOnHost_1(h_cube);
	delete test1;
	hipComplex* d_cube;
	hipMalloc((void**)&d_cube, sizeof(hipComplex)*SIZE);

	hipEventRecord(stop, 0);             //Timing ends
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime1, start, stop);
	outFile << "Create the cube: " << elapsedTime1 << std::endl;
	/* Test 2: Test FFT*/
	float elapsedTime2;
	hipEventRecord(start, 0);

	TestFFTCube_2* test2 = new TestFFTCube_2(h_cube, d_cube, M*L,N);
	delete test2;
	delete[] h_cube;

	hipEventRecord(stop, 0);             //Timing ends
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime2, start, stop);
	outFile << "FFT: " << elapsedTime2 << std::endl;

	/* Test 3: Test steering vector t*/
	float elapsedTime3;
	hipEventRecord(start, 0);

	hipComplex *d_t;
	hipMalloc((void**)&d_t, sizeof(hipComplex)*N*M);

	TestSteeringVector_3 *test3= new TestSteeringVector_3(d_t);
	delete test3;

	hipEventRecord(stop, 0);             //Timing ends
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime3, start, stop);
	outFile << "Create the steering vector: " << elapsedTime3 << std::endl;

	/* Test 4: Test vectorizing slices*/
	float elapsedTime4;
	hipEventRecord(start, 0);

	hipComplex* d_y;
	hipMalloc((void**)&d_y, sizeof(hipComplex)*SIZE);

	TestVectorizeSlices_4 *test4 = new TestVectorizeSlices_4(d_cube, d_y);
	delete test4;
	if (d_cube) hipFree(d_cube);	//free cube

	hipEventRecord(stop, 0);             //Timing ends
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime4, start, stop);
	outFile << "Vectorizing slices: " << elapsedTime4 << std::endl;

	/* Test 5: Compute convariance matrix for every slice*/
	float elapsedTime5;
	hipEventRecord(start, 0);

	hipComplex* d_s;
	hipMalloc((void**)&d_s, sizeof(hipComplex)*M*N*M*N);
	
	
	TestComputeS_5 *test5 = new TestComputeS_5(d_s, d_y);
	delete test5;

	hipEventRecord(stop, 0);             //Timing ends
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime5, start, stop);
	outFile << "Compute convariance matrix: " << elapsedTime5 << std::endl;

	/* Test 6: Get conjugate of t*/
	float elapsedTime6;
	hipEventRecord(start, 0);

	TestConj_6 *test6 = new TestConj_6(d_t);
	delete test6;

	hipEventRecord(stop, 0);             //Timing ends
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime6, start, stop);
	outFile << "Get conjugate of t: " << elapsedTime6 << std::endl;

	/* Test 7: Call Cholesky solver to solve u*/
	float elapsedTime7;
	hipEventRecord(start, 0);

	hipComplex* d_u;
	hipMalloc((void**)&d_u, sizeof(hipComplex)*SIZE);

	TestQR_7 *test7 = new TestQR_7(d_s, M*N, d_t, d_u);	
	delete test7;

	hipEventRecord(stop, 0);             //Timing ends
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime7, start, stop);
	outFile << "Cholesky solver: " << elapsedTime7 << std::endl;

	/* Test 8: Compute t^H * (u*) Note: t is already (t*) in the last step */
	float elapsedTime8;
	//hipEventCreate(&start); //Timing begins
	//hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipComplex* d_tempRes;
	hipMalloc((void**)&d_tempRes, sizeof(hipComplex)*L);

	hipComplex* d_z;
	hipMalloc((void**)&d_z, sizeof(hipComplex)*L);

	TestComputeTU_8 *test8 = new TestComputeTU_8(d_t, d_u, d_tempRes);
	delete test8;

	computeH << <L, M*N >> > (d_u, d_tempRes);	//compute h and store it in d_u
	//matrixScal(d_u, d_tempRes, L);
	//computeZ << <L, M*N >> > (d_u, d_y, d_z);
	matrixBatchMul(d_z, d_u, d_y);
	

	hipEventRecord(stop, 0);             //Timing ends
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime8, start, stop);
	outFile << "Compute z : " << elapsedTime8 << std::endl;

	/*Copy d_z to h_z*/
	float elapsedTime9;
	hipEventRecord(start, 0);

	hipComplex* h_z = (hipComplex*)malloc(L * sizeof(hipComplex));
	hipMemcpy(h_z, d_z, L * sizeof(hipComplex), hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);             //Timing ends
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime9, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	outFile << "Copy z: " << elapsedTime9 << std::endl;

	/*Clean up*/
	if (d_cube) hipFree(d_cube);
	if (d_t) hipFree(d_t);
	if (d_y) hipFree(d_y);
	
	if (d_s) hipFree(d_s);
	if (d_u) hipFree(d_u);
	if (d_z) hipFree(d_z);
	if (h_z) free(h_z);

	hipEventRecord(stop0, 0);             //Timing ends
	hipEventSynchronize(stop0);
	hipEventElapsedTime(&elapsedTime0, start0, stop0);
	hipEventDestroy(start0);
	hipEventDestroy(stop0);

	outFile <<"Total: " << elapsedTime0 << std::endl;
	outFile << "Success!" << std::endl;
	
	//system("pause");
	return 0;
}