#include "hip/hip_runtime.h"
#include "header.cuh"

/*Create a data cube row wisely*/
void createCubeOnHost(hipComplex* h_cube) {
	for (int i = 0; i < L; i++) {
		for (int j = 0; j < M; j++) {
			for (int k = 0; k < N; k++) {
				h_cube[k + j*N + i*M*N] = make_hipComplex(rand(), rand());
			}
		}
	}

}

/*Create steering vector_3*/
/*NxM*/
__global__ void steeringVector(hipComplex* t) {
	int m = threadIdx.x;
	int n = blockIdx.x;
	if (m < M && n < N) {
		hipComplex Fd = make_hipComplex(cosf(n*Fdopp), -sinf(n*Fdopp));
		hipComplex A = make_hipComplex(cosf(m*d*SINTL), -sinf(m*d*SINTL));
		t[n*M + m] = hipCmulf(Fd, A);
	}
}

/*Vectorize Slices _4  Deprecated, using transposition now*/
/*Lx(MxN)*/
__global__ void vectorizeSlices(hipComplex* d_cube, hipComplex* d_y) {
	int sliceNum = blockIdx.x;
	int m = threadIdx.x;
	int n = threadIdx.y;
	int idx_cube = sliceNum*M*N + m*N + n;
	int idx_slice = sliceNum*M*N + n*M + m;

	d_y[idx_slice] = d_cube[idx_cube];
}

/* Matrix Multiplication for hipComplex */
/* C = opA(A) * opB(B)  */
void matrixMulC(hipComplex* d_c, hipComplex* d_y1, hipComplex* d_y2, int m, int n, int k, int lda, int ldb, int ldc, hipblasOperation_t opA, hipblasOperation_t opB) {
	//hipError_t cudaStat; // hipMalloc status 
	hipblasStatus_t stat; // CUBLAS functions status 
	hipblasHandle_t handle; // CUBLAS context 

	stat = hipblasCreate(&handle); // initialize CUBLAS context
	assert(stat == HIPBLAS_STATUS_SUCCESS);
	hipComplex alpha = make_hipComplex(1, 0);
	hipComplex beta = make_hipComplex(0, 0);

	stat = hipblasCgemm(handle, opA, opB, m, n, k,
		&alpha, d_y1, lda, d_y2, ldb, &beta, d_c, ldc);
	assert(stat == HIPBLAS_STATUS_SUCCESS);

	stat = hipblasDestroy(handle); // destroy CUBLAS context
	assert(stat == HIPBLAS_STATUS_SUCCESS);
}

void matrixBatchMul(hipComplex* d_c, hipComplex* d_a, hipComplex* d_b) {
	hipblasStatus_t stat; // CUBLAS functions status 
	hipblasHandle_t handle; // CUBLAS context 

	stat = hipblasCreate(&handle); // initialize CUBLAS context
	assert(stat == HIPBLAS_STATUS_SUCCESS);

	hipComplex alpha = make_hipComplex(1, 0);
	hipComplex beta = make_hipComplex(1, 0);

	stat = hipblasCgemmStridedBatched(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 1, 1, M*N, &alpha, d_a, M*N, M*N, d_b, M*N, M*N, &beta, d_c, L, 1, L);
	assert(stat == HIPBLAS_STATUS_SUCCESS);

	stat = hipblasDestroy(handle); // destroy CUBLAS context
	assert(stat == HIPBLAS_STATUS_SUCCESS);

}

void matrixScal(hipComplex* d_a, hipComplex* d_scaler, int n) {
	hipblasStatus_t stat; // CUBLAS functions status 
	hipblasHandle_t handle; // CUBLAS context 

	stat = hipblasCreate(&handle); // initialize CUBLAS context
	assert(stat == HIPBLAS_STATUS_SUCCESS);

	stat = hipblasCscal(handle, n, d_scaler, d_a, M*N);
	assert(stat == HIPBLAS_STATUS_SUCCESS);

	stat = hipblasDestroy(handle); // destroy CUBLAS context
	assert(stat == HIPBLAS_STATUS_SUCCESS);
}

/*Get conjugate of a vector*/
/*1xMN*/
__global__ void getConj(hipComplex* vector) {
	int i = threadIdx.x;
	vector[i] = make_hipComplex(hipCrealf(vector[i]), -hipCimagf(vector[i]));
}

/*Compute Z _8*/
/*LxMN*/
__global__ void computeTU(hipComplex* d_t, hipComplex* d_u, hipComplex* tempRes) {
	int tid = threadIdx.x;
	int slice = blockIdx.x;
	tempRes[slice] = make_hipComplex(0, 0);
	hipComplex temp = hipCmulf(d_t[slice*M*N + tid], d_u[slice*M*N + tid]);
	tempRes[slice] = hipCaddf(tempRes[slice], temp);
}

/*LxMN*/
__global__ void computeH(hipComplex* d_u, hipComplex* tempRes) {
	int tid = threadIdx.x;
	int sliceNum = blockIdx.x;
	d_u[sliceNum*M*N + tid] = hipCdivf(d_u[sliceNum*M*N + tid], tempRes[sliceNum]); //compute h and store it in d_u
}

/*LxMN*/
__global__ void computeZ(hipComplex* d_u, hipComplex* d_y, hipComplex* d_z) {
	int tid = threadIdx.x;
	int slice = blockIdx.x;
	d_z[slice] = make_hipComplex(0, 0);

	hipComplex temp = hipCmulf(d_u[slice*M*N + tid], d_y[slice*M*N + tid]);
	d_z[slice] = hipCaddf(d_z[slice], temp);
}

void printMatrix(int m, int n, const hipComplex* A, int lda, const char* name)
{
	for (int row = 0; row < m; row++) {
		for (int col = 0; col < n; col++) {
			hipComplex Areg = A[row + col*lda];
			printf("%s(%d,%d) = %f\n", name, row + 1, col + 1, Areg.x);
		}
	}
}