#include "header.cuh"

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

/********/
/* MAIN */
/********/
int batchFFT(hipfftComplex* inputData, hipfftComplex* outputData,int BATCH, int DATASIZE)
{

	// --- Device side input data allocation and initialization
	hipfftComplex *deviceInputData; gpuErrchk(hipMalloc((void**)&deviceInputData, DATASIZE * BATCH * sizeof(hipfftComplex)));
	hipMemcpy(deviceInputData, inputData, DATASIZE * BATCH * sizeof(hipfftComplex), hipMemcpyHostToDevice);
	// --- Batched 1D FFTs
	hipfftHandle handle;
	int rank = 1;                           // --- 1D FFTs
	int n[] = { DATASIZE };                 // --- Size of the Fourier transform
	int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
	int idist = DATASIZE, odist = DATASIZE; // --- Distance between batches
	int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
	int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
	int batch = BATCH;                      // --- Number of batched executions
	hipfftPlanMany(&handle, rank, n,
		inembed, istride, idist,
		onembed, ostride, odist, HIPFFT_C2C, batch);

	//hipfftPlan1d(&handle, DATASIZE, HIPFFT_R2C, BATCH);
	hipfftExecC2C(handle, deviceInputData, outputData, HIPFFT_FORWARD);
	hipfftDestroy(handle);
	

	return 0;
}